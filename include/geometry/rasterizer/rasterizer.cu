#include "hip/hip_runtime.h"
#include "geometry.h"
#include <fstream>
#include <chrono>

#include "cow.h"

static const float inchToMm = 25.4;
enum FitResolutionGate { kFill = 0, kOverscan };

//[comment]
// Compute screen coordinates based on a physically-based camera model
// http://www.scratchapixel.com/lessons/3d-basic-rendering/3d-viewing-pinhole-camera
//[/comment]
void computeScreenCoordinates(
    const float &filmApertureWidth,
    const float &filmApertureHeight,
    const uint32_t &imageWidth,
    const uint32_t &imageHeight,
    const FitResolutionGate &fitFilm,
    const float &nearClippingPLane,
    const float &focalLength,
    float &top, float &bottom, float &left, float &right
)
{
    float filmAspectRatio = filmApertureWidth / filmApertureHeight;
    float deviceAspectRatio = imageWidth / (float)imageHeight;
    
    top = ((filmApertureHeight * inchToMm / 2) / focalLength) * nearClippingPLane;
    right = ((filmApertureWidth * inchToMm / 2) / focalLength) * nearClippingPLane;

    // field of view (horizontal)
    float fov = 2 * 180 / M_PI * atan((filmApertureWidth * inchToMm / 2) / focalLength);
    std::cerr << "Field of view " << fov << std::endl;
    
    float xscale = 1;
    float yscale = 1;
    
    switch (fitFilm) {
        default:
        case kFill:
            if (filmAspectRatio > deviceAspectRatio) {
                xscale = deviceAspectRatio / filmAspectRatio;
            }
            else {
                yscale = filmAspectRatio / deviceAspectRatio;
            }
            break;
        case kOverscan:
            if (filmAspectRatio > deviceAspectRatio) {
                yscale = filmAspectRatio / deviceAspectRatio;
            }
            else {
                xscale = deviceAspectRatio / filmAspectRatio;
            }
            break;
    }
    
    right *= xscale;
    top *= yscale;
    
    bottom = -top;
    left = -right;
}

//[comment]
// Compute vertex raster screen coordinates.
// Vertices are defined in world space. They are then converted to camera space,
// then to NDC space (in the range [-1,1]) and then to raster space.
// The z-coordinates of the vertex in raster space is set with the z-coordinate
// of the vertex in camera space.
//[/comment]
void convertToRaster(
    const Vec3f &vertexWorld,
    const Matrix44f &worldToCamera,
    const float &l,
    const float &r,
    const float &t,
    const float &b,
    const float &near,
    const uint32_t &imageWidth,
    const uint32_t &imageHeight,
    Vec3f &vertexRaster
)
{
    Vec3f vertexCamera;

    worldToCamera.multVecMatrix(vertexWorld, vertexCamera);
    
    // convert to screen space
    Vec2f vertexScreen;
    vertexScreen.x = near * vertexCamera.x / -vertexCamera.z;
    vertexScreen.y = near * vertexCamera.y / -vertexCamera.z;
    
    // now convert point from screen space to NDC space (in range [-1,1])
    Vec2f vertexNDC;
    vertexNDC.x = 2 * vertexScreen.x / (r - l) - (r + l) / (r - l);
    vertexNDC.y = 2 * vertexScreen.y / (t - b) - (t + b) / (t - b);

    // convert to raster space
    vertexRaster.x = (vertexNDC.x + 1) / 2 * imageWidth;
    // in raster space y is down so invert direction
    vertexRaster.y = (1 - vertexNDC.y) / 2 * imageHeight;
    vertexRaster.z = -vertexCamera.z;
}

float min3(const float &a, const float &b, const float &c)
{ return std::min(a, std::min(b, c)); }

float max3(const float &a, const float &b, const float &c)
{ return std::max(a, std::max(b, c)); }

float edgeFunction(const Vec3f &a, const Vec3f &b, const Vec3f &c)
{ return (c[0] - a[0]) * (b[1] - a[1]) - (c[1] - a[1]) * (b[0] - a[0]); }

const uint32_t imageWidth = 640;
const uint32_t imageHeight = 480;
const Matrix44f worldToCamera = {0.707107, -0.331295, 0.624695, 0, 0, 0.883452, 0.468521, 0, -0.707107, -0.331295, 0.624695, 0, -1.63871, -5.747777, -40.400412, 1};

const uint32_t ntris = 3156;
const float nearClippingPLane = 1;
const float farClippingPLane = 1000;
float focalLength = 20; // in mm
// 35mm Full Aperture in inches
float filmApertureWidth = 0.980;
float filmApertureHeight = 0.735;

int main(int argc, char **argv)
{
    Matrix44f cameraToWorld = worldToCamera.inverse();

    // compute screen coordinates
    float t, b, l, r;
    
    computeScreenCoordinates(
        filmApertureWidth, filmApertureHeight,
        imageWidth, imageHeight,
        kOverscan,
        nearClippingPLane,
        focalLength,
        t, b, l, r);
    
    // define the frame-buffer and the depth-buffer. Initialize depth buffer
    // to far clipping plane.
    Vec3<unsigned char> *frameBuffer = new Vec3<unsigned char>[imageWidth * imageHeight];
    for (uint32_t i = 0; i < imageWidth * imageHeight; ++i) frameBuffer[i] = Vec3<unsigned char>(255);
    float *depthBuffer = new float[imageWidth * imageHeight];
    for (uint32_t i = 0; i < imageWidth * imageHeight; ++i) depthBuffer[i] = farClippingPLane;

    auto t_start = std::chrono::high_resolution_clock::now();
    
    // [comment]
    // Outer loop
    // [/comment]
    for (uint32_t i = 0; i < ntris; ++i) {
        const Vec3f &v0 = vertices[nvertices[i * 3]];
        const Vec3f &v1 = vertices[nvertices[i * 3 + 1]];
        const Vec3f &v2 = vertices[nvertices[i * 3 + 2]];
        
        // [comment]
        // Convert the vertices of the triangle to raster space
        // [/comment]
        Vec3f v0Raster, v1Raster, v2Raster;
        convertToRaster(v0, worldToCamera, l, r, t, b, nearClippingPLane, imageWidth, imageHeight, v0Raster);
        convertToRaster(v1, worldToCamera, l, r, t, b, nearClippingPLane, imageWidth, imageHeight, v1Raster);
        convertToRaster(v2, worldToCamera, l, r, t, b, nearClippingPLane, imageWidth, imageHeight, v2Raster);
        
        // [comment]
        // Precompute reciprocal of vertex z-coordinate
        // [/comment]
        v0Raster.z = 1 / v0Raster.z,
        v1Raster.z = 1 / v1Raster.z,
        v2Raster.z = 1 / v2Raster.z;
        
        
        // [comment]
        // Prepare vertex attributes. Divde them by their vertex z-coordinate
        // (though we use a multiplication here because v.z = 1 / v.z)
        // [/comment]
        Vec2f st0 = st[stindices[i * 3]];
        Vec2f st1 = st[stindices[i * 3 + 1]];
        Vec2f st2 = st[stindices[i * 3 + 2]];

        st0 *= v0Raster.z, st1 *= v1Raster.z, st2 *= v2Raster.z;
    
        float xmin = min3(v0Raster.x, v1Raster.x, v2Raster.x);
        float ymin = min3(v0Raster.y, v1Raster.y, v2Raster.y);
        float xmax = max3(v0Raster.x, v1Raster.x, v2Raster.x);
        float ymax = max3(v0Raster.y, v1Raster.y, v2Raster.y);
        
        // the triangle is out of screen
        if (xmin > imageWidth - 1 || xmax < 0 || ymin > imageHeight - 1 || ymax < 0) continue;

        // be careful xmin/xmax/ymin/ymax can be negative. Don't cast to uint32_t
        uint32_t x0 = std::max(int32_t(0), (int32_t)(std::floor(xmin)));
        uint32_t x1 = std::min(int32_t(imageWidth) - 1, (int32_t)(std::floor(xmax)));
        uint32_t y0 = std::max(int32_t(0), (int32_t)(std::floor(ymin)));
        uint32_t y1 = std::min(int32_t(imageHeight) - 1, (int32_t)(std::floor(ymax)));

        float area = edgeFunction(v0Raster, v1Raster, v2Raster);
        
        // [comment]
        // Inner loop
        // [/comment]
        for (uint32_t y = y0; y <= y1; ++y) {
            for (uint32_t x = x0; x <= x1; ++x) {
                Vec3f pixelSample(x + 0.5, y + 0.5, 0);
                float w0 = edgeFunction(v1Raster, v2Raster, pixelSample);
                float w1 = edgeFunction(v2Raster, v0Raster, pixelSample);
                float w2 = edgeFunction(v0Raster, v1Raster, pixelSample);
                if (w0 >= 0 && w1 >= 0 && w2 >= 0) {
                    w0 /= area;
                    w1 /= area;
                    w2 /= area;
                    float oneOverZ = v0Raster.z * w0 + v1Raster.z * w1 + v2Raster.z * w2;
                    float z = 1 / oneOverZ;
                    // [comment]
                    // Depth-buffer test
                    // [/comment]
                    if (z < depthBuffer[y * imageWidth + x]) {
                        depthBuffer[y * imageWidth + x] = z;
                        
                        Vec2f st = st0 * w0 + st1 * w1 + st2 * w2;
                        
                        st *= z;
                        
                        // [comment]
                        // If you need to compute the actual position of the shaded
                        // point in camera space. Proceed like with the other vertex attribute.
                        // Divide the point coordinates by the vertex z-coordinate then
                        // interpolate using barycentric coordinates and finally multiply
                        // by sample depth.
                        // [/comment]
                        Vec3f v0Cam, v1Cam, v2Cam;
                        worldToCamera.multVecMatrix(v0, v0Cam);
                        worldToCamera.multVecMatrix(v1, v1Cam);
                        worldToCamera.multVecMatrix(v2, v2Cam);
                        
                        float px = (v0Cam.x/-v0Cam.z) * w0 + (v1Cam.x/-v1Cam.z) * w1 + (v2Cam.x/-v2Cam.z) * w2;
                        float py = (v0Cam.y/-v0Cam.z) * w0 + (v1Cam.y/-v1Cam.z) * w1 + (v2Cam.y/-v2Cam.z) * w2;
                        
                        Vec3f pt(px * z, py * z, -z); // pt is in camera space
                        
                        // [comment]
                        // Compute the face normal which is used for a simple facing ratio.
                        // Keep in mind that we are doing all calculation in camera space.
                        // Thus the view direction can be computed as the point on the object
                        // in camera space minus Vec3f(0), the position of the camera in camera
                        // space.
                        // [/comment]
                        Vec3f n = (v1Cam - v0Cam).crossProduct(v2Cam - v0Cam);
                        n.normalize();
                        Vec3f viewDirection = -pt;
                        viewDirection.normalize();
                        
                        float nDotView =  std::max(0.f, n.dotProduct(viewDirection));
                        
                        // [comment]
                        // The final color is the reuslt of the faction ration multiplied by the
                        // checkerboard pattern.
                        // [/comment]
                        const int M = 10;
                        float checker = (fmod(st.x * M, 1.0) > 0.5) ^ (fmod(st.y * M, 1.0) < 0.5);
                        float c = 0.3 * (1 - checker) + 0.7 * checker;
                        nDotView *= c;
                        frameBuffer[y * imageWidth + x].x = nDotView * 255;
                        frameBuffer[y * imageWidth + x].y = nDotView * 255;
                        frameBuffer[y * imageWidth + x].z = nDotView * 255;
                    }
                }
            }
        }
    }
    
    auto t_end = std::chrono::high_resolution_clock::now();
	auto passedTime = std::chrono::duration<double, std::milli>(t_end - t_start).count();
	std::cerr << "Wall passed time:  " << passedTime << " ms" << std::endl;
    
    // [comment]
    // Store the result of the framebuffer to a PPM file (Photoshop reads PPM files).
    // [/comment]
    std::ofstream ofs;
    ofs.open("./output.ppm");
    ofs << "P6\n" << imageWidth << " " << imageHeight << "\n255\n";
    ofs.write((char*)frameBuffer, imageWidth * imageWidth * 3);
    ofs.close();
    
    delete [] frameBuffer;
    delete [] depthBuffer;
    
    return 0;
}