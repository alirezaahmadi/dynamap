#include "hip/hip_runtime.h"
#include "utils_Cuda.h"

// Simple Matrix Copy
__global__ void copy(float *odata, const float *idata){
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
        odata[(y+j)*width + x] = idata[(y+j)*width + x];
}

__global__ void copySharedMem(float *odata, const float *idata){
    __shared__ float tile[TILE_DIM * TILE_DIM];

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x] = idata[(y+j)*width + x];

    __syncthreads();

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        odata[(y+j)*width + x] = tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x];          
}
// Naive Matrix Transpose
__global__ void transposeNaive(float *odata, const float *idata){
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
    odata[x*width + (y+j)] = idata[(y+j)*width + x];
}
// Coalesced Transpose Via Shared Memory
__global__ void transposeCoalesced(float *odata, const float *idata, int size){
    int TILE_DIM = size;
    __shared__ float tile[TILE_DIM][TILE_DIM];

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

    __syncthreads();

    x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}
/************************************/
/* SAVE REAL ARRAY FROM GPU TO FILE */
/************************************/
template <class T>
void saveGPUrealtxt(const T * d_in, const char *filename, const int M) {

    T *h_in = (T *)malloc(M * sizeof(T));

    gpuErrchk(hipMemcpy(h_in, d_in, M * sizeof(T), hipMemcpyDeviceToHost));

    std::ofstream outfile;
    outfile.open(filename);
    for (int i = 0; i < M; i++) outfile << std::setprecision(prec_save) << h_in[i] << "\n";
    outfile.close();

}