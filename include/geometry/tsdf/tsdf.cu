#include "hip/hip_runtime.h"
// original implementation Refusion: https://github.com/PRBonn/refusion
#include "tsdf.h"
#include <cfloat>
#include <cmath>
#include "meshExtractor/meshExtractor.h"
#include <unistd.h>
#include <iostream>

namespace DynaMap{

namespace tsdf{

  tsdfVolume::tsdfVolume(void){}
  tsdfVolume::~tsdfVolume(void){
    Free();
  }
  
  void tsdfVolume::init(const tsdfProperties& _Properties){
    Properties = _Properties;
    HashTable::Init(Properties.numBuckets, Properties.bucketSize,
                    Properties.numBlocks, Properties.blockSize);
  }
  void tsdfVolume::Free(){
      HashTable::Free();
  }
  __host__ __device__
  tsdfProperties tsdfVolume::GetProperties(){
    return Properties;
  }
  __device__ 
  float3 tsdfVolume::GlobalVoxelToWorld(int3 position) { // todo ..
    float3 result = make_float3(position.x * Properties.voxelSize,
                                position.y * Properties.voxelSize,
                                position.z * Properties.voxelSize);
    return result;
  }
  __device__ 
  float3 tsdfVolume::voxelArrayIndexToWorld(int index) { // todo ...
    int m = GetProperties().voxelNumPerAxes;
    int n = m*m;
    float3 result = GlobalVoxelToWorld(make_int3(__float2uint_rd(index%m), 
                                                 __float2uint_rd((index%n)/m), 
                                                 __float2uint_rd(index/n)));
    return result; 
  }
  __device__
  geometry::Voxel tsdfVolume::GetInterpolatedVoxel(float3 position) {
    geometry::Voxel v0 = GetVoxel(position);
    if (v0.weight == 0) return v0;
    float voxel_size = Properties.voxelSize;
    const float3 pos_dual =
        position -
        make_float3(voxel_size / 2.0f, voxel_size / 2.0f, voxel_size / 2.0f);
    float3 voxel_position = position / voxel_size;
    float3 weight = make_float3(voxel_position.x - floor(voxel_position.x),
                                voxel_position.y - floor(voxel_position.y),
                                voxel_position.z - floor(voxel_position.z));
  
    float distance = 0.0f;
    float3 color_float = make_float3(0.0f, 0.0f, 0.0f);
    float3 vColor;
  
    geometry::Voxel v = GetVoxel(pos_dual + make_float3(0.0f, 0.0f, 0.0f));
    if (v.weight == 0) {
      vColor = make_float3(v0.color.x, v0.color.y, v0.color.z);
      distance +=
          (1.0f - weight.x) * (1.0f - weight.y) * (1.0f - weight.z) * v0.sdf;
      color_float =
          color_float +
          (1.0f - weight.x) * (1.0f - weight.y) * (1.0f - weight.z) * vColor;
    } else {
      vColor = make_float3(v.color.x, v.color.y, v.color.z);
      distance +=
          (1.0f - weight.x) * (1.0f - weight.y) * (1.0f - weight.z) * v.sdf;
      color_float =
          color_float +
          (1.0f - weight.x) * (1.0f - weight.y) * (1.0f - weight.z) * vColor;
    }
  
    v = GetVoxel(pos_dual + make_float3(voxel_size, 0.0f, 0.0f));
    if (v.weight == 0) {
      vColor = make_float3(v0.color.x, v0.color.y, v0.color.z);
      distance += weight.x * (1.0f - weight.y) * (1.0f - weight.z) * v0.sdf;
      color_float =
          color_float + weight.x * (1.0f - weight.y) * (1.0f - weight.z) * vColor;
    } else {
      vColor = make_float3(v.color.x, v.color.y, v.color.z);
      distance += weight.x * (1.0f - weight.y) * (1.0f - weight.z) * v.sdf;
      color_float =
          color_float + weight.x * (1.0f - weight.y) * (1.0f - weight.z) * vColor;
    }
  
    v = GetVoxel(pos_dual + make_float3(0.0f, voxel_size, 0.0f));
    if (v.weight == 0) {
      vColor = make_float3(v0.color.x, v0.color.y, v0.color.z);
      distance += (1.0f - weight.x) * weight.y * (1.0f - weight.z) * v0.sdf;
      color_float =
          color_float + (1.0f - weight.x) * weight.y * (1.0f - weight.z) * vColor;
    } else {
      vColor = make_float3(v.color.x, v.color.y, v.color.z);
      distance += (1.0f - weight.x) * weight.y * (1.0f - weight.z) * v.sdf;
      color_float =
          color_float + (1.0f - weight.x) * weight.y * (1.0f - weight.z) * vColor;
    }
  
    v = GetVoxel(pos_dual + make_float3(0.0f, 0.0f, voxel_size));
    if (v.weight == 0) {
      vColor = make_float3(v0.color.x, v0.color.y, v0.color.z);
      distance += (1.0f - weight.x) * (1.0f - weight.y) * weight.z * v0.sdf;
      color_float =
          color_float + (1.0f - weight.x) * (1.0f - weight.y) * weight.z * vColor;
    } else {
      vColor = make_float3(v.color.x, v.color.y, v.color.z);
      distance += (1.0f - weight.x) * (1.0f - weight.y) * weight.z * v.sdf;
      color_float =
          color_float + (1.0f - weight.x) * (1.0f - weight.y) * weight.z * vColor;
    }
  
    v = GetVoxel(pos_dual + make_float3(voxel_size, voxel_size, 0.0f));
    if (v.weight == 0) {
      vColor = make_float3(v0.color.x, v0.color.y, v0.color.z);
      distance += weight.x * weight.y * (1.0f - weight.z) * v0.sdf;
      color_float =
          color_float + weight.x * weight.y * (1.0f - weight.z) * vColor;
    } else {
      vColor = make_float3(v.color.x, v.color.y, v.color.z);
      distance += weight.x * weight.y * (1.0f - weight.z) * v.sdf;
      color_float =
          color_float + weight.x * weight.y * (1.0f - weight.z) * vColor;
    }
  
    v = GetVoxel(pos_dual + make_float3(0.0f, voxel_size, voxel_size));
    if (v.weight == 0) {
      vColor = make_float3(v0.color.x, v0.color.y, v0.color.z);
      distance += (1.0f - weight.x) * weight.y * weight.z * v0.sdf;
      color_float =
          color_float + (1.0f - weight.x) * weight.y * weight.z * vColor;
    } else {
      vColor = make_float3(v.color.x, v.color.y, v.color.z);
      distance += (1.0f - weight.x) * weight.y * weight.z * v.sdf;
      color_float =
          color_float + (1.0f - weight.x) * weight.y * weight.z * vColor;
    }
  
    v = GetVoxel(pos_dual + make_float3(voxel_size, 0.0f, voxel_size));
    if (v.weight == 0) {
      vColor = make_float3(v0.color.x, v0.color.y, v0.color.z);
      distance += weight.x * (1.0f - weight.y) * weight.z * v0.sdf;
      color_float =
          color_float + weight.x * (1.0f - weight.y) * weight.z * vColor;
    } else {
      vColor = make_float3(v.color.x, v.color.y, v.color.z);
      distance += weight.x * (1.0f - weight.y) * weight.z * v.sdf;
      color_float =
          color_float + weight.x * (1.0f - weight.y) * weight.z * vColor;
    }
  
    v = GetVoxel(pos_dual + make_float3(voxel_size, voxel_size, voxel_size));
    if (v.weight == 0) {
      vColor = make_float3(v0.color.x, v0.color.y, v0.color.z);
      distance += weight.x * weight.y * weight.z * v0.sdf;
      color_float = color_float + weight.x * weight.y * weight.z * vColor;
  
    } else {
      vColor = make_float3(v.color.x, v.color.y, v.color.z);
      distance += weight.x * weight.y * weight.z * v.sdf;
      color_float = color_float + weight.x * weight.y * weight.z * vColor;
    }
  
    uchar3 color = make_uchar3(color_float.x, color_float.y, color_float.z);
    v.weight = v0.weight;
    v.sdf = distance;
    v.color = color;
    return v;
  }
  __device__
  int tsdfVolume::WorldToGlobalVoxel(float3 position) {
    int m = Properties.voxelNumPerAxes;
    int3 poseIndexs =  make_int3(position.x / Properties.voxelSize,
                                 position.y / Properties.voxelSize,
                                 position.z / Properties.voxelSize);

    // printf("x: %i, y: %i, z: %i \n",position.x, position.y,position.z);
    return __float2int_rd(poseIndexs.x +
            poseIndexs.y * m + 
            poseIndexs.z * m * m);
  }
  __device__
  int3 tsdfVolume::WorldToGlobalVoxelHashing(float3 position) {
    return make_int3(position.x / Properties.voxelSize + signf(position.x) * 0.5f,
                     position.y / Properties.voxelSize + signf(position.y) * 0.5f,
                     position.z / Properties.voxelSize + signf(position.z) * 0.5f);
  }
  __device__ 
  int3 tsdfVolume::WorldToBlock(float3 position) {
    int3 voxel_position = WorldToGlobalVoxelHashing(position);
    int3 block_position;
    if (voxel_position.x < 0)
      block_position.x = (voxel_position.x - block_size_ + 1) / block_size_;
    else
      block_position.x = voxel_position.x / block_size_;
  
    if (voxel_position.y < 0)
      block_position.y = (voxel_position.y - block_size_ + 1) / block_size_;
    else
      block_position.y = voxel_position.y / block_size_;
  
    if (voxel_position.z < 0)
      block_position.z = (voxel_position.z - block_size_ + 1) / block_size_;
    else
      block_position.z = voxel_position.z / block_size_;
  
    return block_position;
  }
  __device__ 
  int3 tsdfVolume::WorldToLocalVoxel(float3 position) {
    int3 position_global = WorldToGlobalVoxelHashing(position);
    int3 position_local = make_int3(position_global.x % block_size_,
                                    position_global.y % block_size_,
                                    position_global.z % block_size_);
    if (position_local.x < 0) position_local.x += block_size_;
    if (position_local.y < 0) position_local.y += block_size_;
    if (position_local.z < 0) position_local.z += block_size_;
    return position_local;
  }
  __device__ 
  geometry::Voxel tsdfVolume::GetVoxel(float3 position) {
    int3 block_position = WorldToBlock(position);
    int3 local_voxel = WorldToLocalVoxel(position);
    HashEntry entry = HashTable::FindHashEntry(block_position);
    if (entry.pointer == kFreeEntry) {
      geometry::Voxel voxel;
      voxel.sdf = 0;
      voxel.color = make_uchar3(0, 0, 0);
      voxel.weight = 0;
      return voxel;
    }
    return HashTable::voxel_blocks_[entry.pointer].at(local_voxel);
  }
  __device__ 
  bool tsdfVolume::SetVoxel(float3 position, const geometry::Voxel& voxel) {
    int3 block_position = WorldToBlock(position);
    int3 local_voxel = WorldToLocalVoxel(position);
    HashEntry entry = HashTable::FindHashEntry(block_position);
    if (entry.pointer == kFreeEntry) {
      return false;
    }
    HashTable::voxel_blocks_[entry.pointer].at(local_voxel) = voxel;
    return true;
  }
  __device__ 
  bool tsdfVolume::UpdateVoxel(float3 position, const geometry::Voxel& voxel) {
    int3 block_position = WorldToBlock(position);
    int3 local_voxel = WorldToLocalVoxel(position);
    HashEntry entry = HashTable::FindHashEntry(block_position);
    if (entry.pointer == kFreeEntry) {
      return false;
    }
    HashTable::voxel_blocks_[entry.pointer]
        .at(local_voxel)
        .mergeVoxel(voxel, Properties.maxSDFWeight);
        // printf("sdf: %f, W: %i \n",voxel_blocks_[entry.pointer]
        // .at(local_voxel).sdf, voxel_blocks_[entry.pointer]
        // .at(local_voxel).weight);
    return true;
  }
  __global__ 
  void IntegrateScanKernel(tsdfVolume *volume, uchar3 *color,
                                    float *depth, rgbdSensor sensor,
                                    float4x4 transform, float4x4 invTransform) {
    //loop through ALL entries
    //  if entry is in camera frustum
    //    loop through voxels inside block
      //    if voxel is in truncation region
      //      update voxels
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    int block_size = volume->GetProperties().blockSize;
    float voxel_size = volume->GetProperties().voxelSize;
    float truncation_distance = volume->GetProperties().truncationDistance;

    for (int i = index; i < volume->GetNumEntries(); i += stride) {
      float3 position = make_float3(
          volume->GetHashEntry(i).position.x * voxel_size * block_size,
          volume->GetHashEntry(i).position.y * voxel_size * block_size,
          volume->GetHashEntry(i).position.z * voxel_size * block_size);
      // To camera coordinates
      float3 position_cam = invTransform * position;
      // If behind camera plane discard
      if (position_cam.z < 0) continue;
      float3 block_center =
          make_float3(position_cam.x + 0.5 * voxel_size * block_size,
                      position_cam.y + 0.5 * voxel_size * block_size,
                      position_cam.z + 0.5 * voxel_size * block_size);
      int2 image_position = Project(block_center, sensor);
      if (image_position.x >= 0 && image_position.y >= 0 &&
          image_position.x < sensor.cols && image_position.y < sensor.rows) {
        float3 start_pt = make_float3(0, 0, 0);

        for (int bx = 0; bx < block_size; bx++) {
          for (int by = 0; by < block_size; by++) {
            for (int bz = 0; bz < block_size; bz++) {
              float3 voxel_position = make_float3(position.x + bx * voxel_size,
                                                  position.y + by * voxel_size,
                                                  position.z + bz * voxel_size);
              voxel_position = invTransform * voxel_position;
              image_position = Project(voxel_position, sensor);
              // Check again inside the block
              if (image_position.x >= 0 && image_position.y >= 0 &&
                  image_position.x < sensor.cols &&
                  image_position.y < sensor.rows) {
                int idx = image_position.y * sensor.cols + image_position.x;
                if (depth[idx] < volume->GetProperties().minSensorDepth) continue;
                if (depth[idx] > volume->GetProperties().maxSensorDepth) continue;
                float3 point3d = getPoint3d(idx, depth[idx], sensor);
                float surface_distance = distance(start_pt, point3d);
                float voxel_distance = distance(start_pt, voxel_position);
                if (voxel_distance > surface_distance - truncation_distance &&
                    voxel_distance < surface_distance + truncation_distance &&
                    (depth[idx] < volume->GetProperties().maxSensorDepth)) {
                  geometry::Voxel voxel;
                  voxel.sdf = surface_distance - voxel_distance;
                  voxel.color = color[idx];
                  voxel.weight = (unsigned char)1;
                  // To world coordinates
                  voxel_position = transform * voxel_position;
                  volume->UpdateVoxel(voxel_position, voxel);
                } else if (voxel_distance <
                          surface_distance - truncation_distance) {
                  voxel_position = transform * voxel_position;
                  geometry::Voxel voxel;
                  voxel.sdf = truncation_distance;
                  voxel.color = color[idx];
                  voxel.weight = (unsigned char)1;
                  volume->UpdateVoxel(voxel_position, voxel);
                }
              }
            }
          }
        }  // End single block update
      }
    }
  }
  __global__ 
  void AllocateFromDepthKernel(tsdfVolume *volume, float *depth,
                                        rgbdSensor sensor, float4x4 transform) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int size = sensor.rows * sensor.cols;

    float truncation_distance = volume->GetProperties().truncationDistance;
    float block_size =
        volume->GetProperties().blockSize * volume->GetProperties().voxelSize;

    float3 start_pt = make_float3(transform.m14, transform.m24, transform.m34);
    for (int i = index; i < size; i += stride) {
      if (depth[i] < volume->GetProperties().minSensorDepth ||
          depth[i] > volume->GetProperties().maxSensorDepth)
        continue;
      float3 point = getPoint3d(i, depth[i], sensor);
      point = transform * point;
      if (point.x == 0 && point.y == 0 && point.z == 0) continue;
      // compute start and end of the ray
      float3 ray_direction = normalize(point - start_pt);
      float surface_distance = distance(start_pt, point);
      float3 ray_start = start_pt;
      float3 ray_end =
          start_pt + ray_direction * (surface_distance + truncation_distance);
      // traverse the ray discretely using the block size and allocate it
      // adapted from https://github.com/francisengelmann/fast_voxel_traversal/blob/master/main.cpp
      int3 block_start = make_int3(floor(ray_start.x / block_size),
                                  floor(ray_start.y / block_size),
                                  floor(ray_start.z / block_size));

      int3 block_end = make_int3(floor(ray_end.x / block_size),
                                floor(ray_end.y / block_size),
                                floor(ray_end.z / block_size));

      int3 block_position = block_start;
      int3 step = make_int3(sign(ray_direction.x),
                            sign(ray_direction.y),
                            sign(ray_direction.z));

      float3 delta_t;
      delta_t.x =
          (ray_direction.x != 0) ? fabs(block_size / ray_direction.x) : FLT_MAX;
      delta_t.y =
          (ray_direction.y != 0) ? fabs(block_size / ray_direction.y) : FLT_MAX;
      delta_t.z =
          (ray_direction.z != 0) ? fabs(block_size / ray_direction.z) : FLT_MAX;

      float3 boundary = make_float3(
          (block_position.x + static_cast<float>(step.x)) * block_size,
          (block_position.y + static_cast<float>(step.y)) * block_size,
          (block_position.z + static_cast<float>(step.z)) * block_size);

      float3 max_t;
      max_t.x = (ray_direction.x != 0)
                    ? (boundary.x - ray_start.x) / ray_direction.x
                    : FLT_MAX;
      max_t.y = (ray_direction.y != 0)
                    ? (boundary.y - ray_start.y) / ray_direction.y
                    : FLT_MAX;
      max_t.z = (ray_direction.z != 0)
                    ? (boundary.z - ray_start.z) / ray_direction.z
                    : FLT_MAX;

      int3 diff = make_int3(0, 0, 0);
      bool neg_ray = false;

      if (block_position.x != block_end.x && ray_direction.x < 0) {
        diff.x--;
        neg_ray = true;
      }
      if (block_position.y != block_end.y && ray_direction.y < 0) {
        diff.y--;
        neg_ray = true;
      }
      if (block_position.z != block_end.z && ray_direction.z < 0) {
        diff.z--;
        neg_ray = true;
      }
      volume->AllocateBlock(block_position);

      if (neg_ray) {
        block_position = block_position + diff;
        volume->AllocateBlock(block_position);
      }

      while (block_position.x != block_end.x || block_position.y != block_end.y ||
            block_position.z != block_end.z) {
        if (max_t.x < max_t.y) {
          if (max_t.x < max_t.z) {
            block_position.x += step.x;
            max_t.x += delta_t.x;
          } else {
            block_position.z += step.z;
            max_t.z += delta_t.z;
          }
        } else {
          if (max_t.y < max_t.z) {
            block_position.y += step.y;
            max_t.y += delta_t.y;
          } else {
            block_position.z += step.z;
            max_t.z += delta_t.z;
          }
        }
        volume->AllocateBlock(block_position);
      }
    }
  }
  void tsdfVolume::IntegrateScan(const rgbdImage &image, float4x4 camera_pose) {
      int threads_per_block = 64;
      int thread_blocks =
          (Properties.numBuckets * Properties.bucketSize + threads_per_block - 1) /
          threads_per_block;

      AllocateFromDepthKernel<<<thread_blocks, threads_per_block>>>(
          this, image.depth, image.sensor, camera_pose);
      hipDeviceSynchronize();
      
      float4x4 inv_camera_pose = camera_pose.getInverse();
      IntegrateScanKernel<<<thread_blocks, threads_per_block>>>(
          this, image.rgb, image.depth, image.sensor, camera_pose,
          inv_camera_pose);
      hipDeviceSynchronize();
  }
  __global__ 
  void GenerateDepthKernel(tsdfVolume *volume, rgbdSensor sensor,
                           float4x4 camera_pose,
                           float *virtual_depth) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int size = sensor.rows * sensor.cols;

    float3 start_pt = make_float3(camera_pose.m14, camera_pose.m24, camera_pose.m34);
    for (int i = index; i < size; i += stride) {
      float current_depth = 0;
      while (current_depth < volume->GetProperties().maxSensorDepth) {
        float3 point = getPoint3d(i, current_depth, sensor);
        point = camera_pose * point;
        geometry::Voxel v = volume->GetInterpolatedVoxel(point);
        if (v.weight == 0) {
          current_depth += volume->GetProperties().truncationDistance;
        } else {
          current_depth += v.sdf;
        }
        if (v.weight != 0 && v.sdf < volume->GetProperties().voxelSize)break;
      }
      virtual_depth[i] = current_depth;
    }
  }
  __global__ 
  void GenerateRgbKernel(tsdfVolume *volume, rgbdSensor sensor,
                                  float4x4 camera_pose, uchar3 *virtual_rgb) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int size = sensor.rows * sensor.cols;

    float3 start_pt =
    make_float3(camera_pose.m14, camera_pose.m24, camera_pose.m34);
    for (int i = index; i < size; i += stride) {
      float current_depth = 0;
      while (current_depth < volume->GetProperties().maxSensorDepth) {
      float3 point = getPoint3d(i, current_depth, sensor);
      point = camera_pose * point;
      geometry::Voxel v = volume->GetInterpolatedVoxel(point);
      if (v.weight == 0) {
        current_depth += volume->GetProperties().truncationDistance;
      } else {
        current_depth += v.sdf;
      }
      if (v.weight != 0 && v.sdf < volume->GetProperties().voxelSize) break;
      }
      if (current_depth < volume->GetProperties().maxSensorDepth) {
        float3 point = getPoint3d(i, current_depth, sensor);
        point = camera_pose * point;
        geometry:: Voxel v = volume->GetInterpolatedVoxel(point);
        virtual_rgb[i] = v.color;
      } else {
        virtual_rgb[i] = make_uchar3(0, 0, 0);
      }
    }
  }
  float* tsdfVolume::GenerateDepth(float4x4 camera_pose, rgbdSensor sensor) {
    float* virtual_depth;
    hipMallocManaged(&virtual_depth, sizeof(float) * sensor.rows * sensor.cols);
    int threads_per_block = 64;
    int thread_blocks =
    (sensor.rows * sensor.cols + threads_per_block - 1) / threads_per_block;
    GenerateDepthKernel<<<thread_blocks, threads_per_block>>>(
    this, sensor, camera_pose, virtual_depth);
    hipDeviceSynchronize();
    // 
    return virtual_depth;
  }
  uchar3* tsdfVolume::GenerateRgb(float4x4 camera_pose, rgbdSensor sensor) {
    uchar3* virtual_rgb;
    hipMallocManaged(&virtual_rgb, sizeof(float) * sensor.rows * sensor.cols);
    int threads_per_block = 64;
    int thread_blocks =
    (sensor.rows * sensor.cols + threads_per_block - 1) / threads_per_block;
    GenerateRgbKernel<<<thread_blocks, threads_per_block>>>(
    this, sensor, camera_pose, virtual_rgb);
    hipDeviceSynchronize();
    
    return virtual_rgb;
  }
  geometry::Mesh tsdfVolume::ExtractMesh(const float3 &lower_corner, const float3 &upper_corner) {
    geometry::MeshExtractor *mesh_extractor;
    hipMallocManaged(&mesh_extractor, sizeof(geometry::MeshExtractor));
    mesh_extractor->Init(200000, Properties.voxelSize);
    mesh_extractor->ExtractMesh(this, lower_corner, upper_corner);
    geometry::Mesh *mesh;
    hipMallocManaged(&mesh, sizeof(geometry::Mesh));
    *mesh = mesh_extractor->GetMesh();
    return *mesh;
  }

}  // namespace tsdf
}  // namespace DynaMap