#include "hip/hip_runtime.h"
// Copyright 2019 Emanuele Palazzolo (emanuele.palazzolo@uni-bonn.de), Cyrill Stachniss, University of Bonn
#include "mesh.h"
#include <algorithm>

namespace DynaMap {
namespace geometry {

void Mesh::Init(unsigned int max_triangles) {
  num_triangles_ = 0;
  max_triangles_ = max_triangles;
  hipMallocManaged(&triangles_, sizeof(Triangle) * max_triangles_);
  hipDeviceSynchronize();
}

void Mesh::Free() {
  hipDeviceSynchronize();
  hipFree(triangles_);
}

__device__ void Mesh::AppendTriangle(Triangle t) {
  unsigned int idx = atomicAdd(&num_triangles_, 1);
  if (num_triangles_ <= max_triangles_) triangles_[idx] = t;
}

__device__ Vertex Mesh::getVertices(unsigned int id){
  Vertex result;
  switch(id%3){
    case 0:{
      result = triangles_[id/3].v0;
    }case 1:{
      result = triangles_[id/3].v1;
    }case 2:{
      result = triangles_[id/3].v2;
    }
  }
  return result;
}

int Mesh::SaveToFile(const std::string &filename) {
  std::ofstream fout(filename.c_str(), std::ofstream::out);
  int n = std::min(num_triangles_, max_triangles_);
  if (n == max_triangles_) {
    std::cout << "Triangles limit reached!" << std::endl;
  }
  for (unsigned int i = 0; i < n; i++) {

    // std::cout << "v " << triangles_[i].v0.position.x << " "
    // << triangles_[i].v0.position.y << " " << triangles_[i].v0.position.z
    // << " " << triangles_[i].v0.color.x << " " << triangles_[i].v0.color.y
    // << " " << triangles_[i].v0.color.z << std::endl;

    fout << "v " << triangles_[i].v0.position.x << " "
         << triangles_[i].v0.position.y << " " << triangles_[i].v0.position.z
         << " " << triangles_[i].v0.color.x << " " << triangles_[i].v0.color.y
         << " " << triangles_[i].v0.color.z << std::endl;
    fout << "v " << triangles_[i].v1.position.x << " "
         << triangles_[i].v1.position.y << " " << triangles_[i].v1.position.z
         << " " << triangles_[i].v1.color.x << " " << triangles_[i].v1.color.y
         << " " << triangles_[i].v1.color.z << std::endl;
    fout << "v " << triangles_[i].v2.position.x << " "
         << triangles_[i].v2.position.y << " " << triangles_[i].v2.position.z
         << " " << triangles_[i].v2.color.x << " " << triangles_[i].v2.color.y
         << " " << triangles_[i].v2.color.z << std::endl;
  }
  for (unsigned int i = 1; i <= n * 3; i += 3) {
    fout << "f " << i << " " << i + 1 << " " << i + 2 << std::endl;
  }
  fout.close();

  return num_triangles_;
}

}  // namespace geometry

}  // namespace DynaMap
